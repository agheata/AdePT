// SPDX-FileCopyrightText: 2020 CERN
// SPDX-License-Identifier: Apache-2.0

/**
 * @file test_atomic.cu
 * @brief Unit test for atomic operations.
 * @author Andrei Gheata (andrei.gheata@cern.ch)
 */

#include <iostream>
#include <cassert>
#include <AdePT/Atomic.h>

#include "test_atomic.h"

///______________________________________________________________________________________
int main(void)
{
  const char *result[2] = {"FAILED", "OK"};
  bool success          = true;
  // Define the kernels granularity: 10K blocks of 32 treads each
  dim3 nblocks(10000), nthreads(32);

  // Allocate the content of SomeStruct in a buffer
  char *buffer = nullptr;
  hipMallocManaged((void**)&buffer, sizeof(SomeStruct));
  SomeStruct *a = SomeStruct::MakeInstanceAt(buffer);

  // Launch a kernel doing additions
  bool testOK = true;
  std::cout << "   testAdd ... ";
  // Wait memory to reach device
  hipDeviceSynchronize();
  #pragma omp parallel for collapse(2)
  COPCORE_KERNEL(nblocks.x, nthreads.x, testAdd, a);
  // Wait all warps to finish and sync memory
  hipDeviceSynchronize();

  testOK &= a->var_int.load() == nblocks.x * nthreads.x;
  testOK &= a->var_float.load() == float(nblocks.x * nthreads.x);
  std::cout << result[testOK] << "\n";
  success &= testOK;

  // Launch a kernel doing subtraction
  testOK = true;
  std::cout << "   testSub ... ";
  a->var_int.store(nblocks.x * nthreads.x);
  a->var_float.store(nblocks.x * nthreads.x);
  hipDeviceSynchronize();
  #pragma omp parallel for collapse(2)
  COPCORE_KERNEL(nblocks.x, nthreads.x, testSub, a);
  hipDeviceSynchronize();

  testOK &= a->var_int.load() == 0;
  testOK &= a->var_float.load() == 0;
  std::cout << result[testOK] << "\n";
  success &= testOK;

  // Launch a kernel testing compare and swap operations
  std::cout << "   testCAS ... ";
  a->var_int.store(99);
  hipDeviceSynchronize();
  #pragma omp parallel for collapse(2)
  COPCORE_KERNEL(nblocks.x, nthreads.x, testCompareExchange, a);
  hipDeviceSynchronize();
  testOK = a->var_int.load() == 99;
  std::cout << result[testOK] << "\n";
  success &= testOK;

  hipFree(buffer);
  if (!success) return 1;
  return 0;
}
